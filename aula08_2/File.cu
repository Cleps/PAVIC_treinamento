#include "hip/hip_runtime.h"

#include "include/hip/hip_runtime.h"
#include "include/"


#include <stdio.h>
#include "File.h"

// Device code  - GPU 
__global__ void HelloGPU(void) {
	printf("  Hello CUDA GPU\n");
}
int main() {

	printf("  Hello CPU 01 \n");

	HelloGPU << < 1, 1 >> > (); // Call GPU

	printf("  Hello CPU 02 \n");
	return 0;
}