#include "hip/hip_runtime.h"
//CUDA - Images - Color to Grayscale -  GPU


//Load Images
#define STB_IMAGE_IMPLEMENTATION
// Write Images
#define STB_IMAGE_WRITE_IMPLEMENTATION

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

#include <iostream>
#include <string>
#include <cassert>

#include "include/stb_image.h"
#include "include/stb_image_write.h"

struct Pixel
{
    unsigned char r, g, b, a;
};

// Cuda Image processing Color to GrayScale - CPU
void ConvertImageToGrayCpu(unsigned char* imageRGBA, int width, int height)
{
    for (int y = 0; y < height; y++)
    {
        for (int x = 0; x < width; x++)
        {
            Pixel* ptrPixel = (Pixel*)&imageRGBA[y * width * 4 + 4 * x];
            unsigned char pixelValue = (unsigned char)(ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
            ptrPixel->r = pixelValue;
            ptrPixel->g = pixelValue;
            ptrPixel->b = pixelValue;
            ptrPixel->a = 255;
        }
    }
}

// Cuda Image processing Color to GrayScale - GPU
__global__ void ConvertImageToGrayGpu(unsigned char* imageRGBA)
{
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t idx = y * blockDim.x * gridDim.x + x;

    Pixel* ptrPixel = (Pixel*)&imageRGBA[idx * 4];
    unsigned char pixelValue = (unsigned char)
        (ptrPixel->r * 0.2126f + ptrPixel->g * 0.7152f + ptrPixel->b * 0.0722f);
    ptrPixel->r = pixelValue;
    ptrPixel->g = pixelValue;
    ptrPixel->b = pixelValue;
    ptrPixel->a = 255;
}


char* filename_01 = "ship_4k_rgba.png";
char* filename_02 = "apple.jpg";

int main()
{


    // Open image
    int width, height, componentCount;
    std::cout << "Loading png file...";
    //unsigned char* imageData = stbi_load(argv[1], &width, &height, &componentCount, 4);
    //unsigned char* imageData = stbi_load("ship_4k_rgba.png", &width, &height, &componentCount, 4);
    unsigned char* imageData = stbi_load(filename_01, &width, &height, &componentCount, 4);
    std::cout << " DONE" << std::endl;

    if (!imageData)
    {
        //std::cout << std::endl << "Failed to open \"" << argv[1] << "\"";
        std::cout << std::endl << "Failed to open \"" << imageData << "\"";
        return -1;
    }
    std::cout << " DONE" << std::endl;

    // Validate image sizes
    if (width % 32 || height % 32)
    {
        // NOTE: Leaked memory of "imageData"
        std::cout << "Width and/or Height is not dividable by 32!";
        return -1;
    }

    /*
    // Process image on cpu
    std::cout << "Processing image...";
    ConvertImageToGrayCpu(imageData, width, height);
    std::cout << " DONE" << std::endl;
    */

    // Copy data to the gpu
    std::cout << "Copy data to GPU...";
    unsigned char* ptrImageDataGpu = nullptr;
    assert(hipMalloc(&ptrImageDataGpu, width * height * 4) == hipSuccess);
    assert(hipMemcpy(ptrImageDataGpu, imageData, width * height * 4, hipMemcpyHostToDevice) == hipSuccess);
    std::cout << " DONE" << std::endl;

    // Process image on gpu
    std::cout << "Running CUDA Kernel...";
    dim3 blockSize(32, 32);
    dim3 gridSize(width / blockSize.x, height / blockSize.y);
    //ConvertImageToGrayGpu <<<gridSize, blockSize >>> (ptrImageDataGpu);
    ConvertImageToGrayGpu << <gridSize, blockSize >> > (ptrImageDataGpu);

    auto err = hipGetLastError();
    std::cout << " DONE" << std::endl;

    // Copy data from the gpu
    std::cout << "Copy data from GPU...";
    assert(hipMemcpy(imageData, ptrImageDataGpu, width * height * 4, hipMemcpyDeviceToHost) == hipSuccess);
    std::cout << " DONE" << std::endl;

    // Build output filename
    std::string fileNameOut = argv[1];
    fileNameOut = fileNameOut.substr(0, fileNameOut.find_last_of('.')) + "_gray.png";

    // Write image back to disk
    std::cout << "Writing png to disk...";
    stbi_write_png(fileNameOut.c_str(), width, height, 4, imageData, 4 * width);
    std::cout << " DONE";

    // Free memory
    hipFree(ptrImageDataGpu);
    stbi_image_free(imageData);
}